
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <iostream>

// TODO: annotate this function with __host__ __device__ so
//       so that they are able to work with Thrust
float2 operator+(float2 a, float2 b)
{
  return make_float2(a.x + b.x, a.y + b.y);
}


void generate_random_points(std::vector<float2> &points)
{
  // sequentially generate some random 2D points in the unit square
  // TODO: parallelize this loop using thrust::tabulate and thrust::default_random_engine
  for(int i = 0; i < points.size(); ++i)
  {
    float x = float(rand()) / RAND_MAX;
    float y = float(rand()) / RAND_MAX;

    points[i] = make_float2(x,y);
  }
}


float2 compute_centroid(const std::vector<float2> &points)
{
  float2 result = make_float2(0,0);

  // compute the mean
  // TODO: parallelize this sum using thrust::reduce
  for(int i = 0; i < points.size(); ++i)
  {
    result = result + points[i];
  }

  return make_float2(result.x / points.size(), result.y / points.size());
}


void classify(const std::vector<float2> &points, float2 centroid, std::vector<int> &quadrants)
{
  // classify each point relative to the centroid
  // TODO: parallelize this loop using thrust::transform
  for(int i = 0; i < points.size(); ++i)
  {
    float x = points[i].x;
    float y = points[i].y;

    // bottom-left:  0
    // bottom-right: 1
    // top-left:     2
    // top-right:    3

    quadrants[i] = (x <= centroid.x ? 0 : 1) | (y <= centroid.y ? 0 : 2);
  }
}


void count_points_in_quadrants(std::vector<float2> &points, std::vector<int> &quadrants, std::vector<int> &counts_per_quadrant)
{
  // sequentially compute a histogram
  // TODO: parallelize this operation by
  //   1. sorting points by quadrant
  //   2. reducing points by quadrant
  for(int i = 0; i < quadrants.size(); ++i)
  {
    int q = quadrants[i];

    // increment the number of points in this quadrant
    counts_per_quadrant[q]++;
  }
}


std::ostream &operator<<(std::ostream &os, float2 p)
{
  return os << "(" << p.x << ", " << p.y << ")";
}


int main()
{
  const size_t num_points = 10;

  // TODO move these points to the GPU by using thrust::device_vector
  std::vector<float2> points(num_points);

  generate_random_points(points);

  for(int i = 0; i < points.size(); ++i)
    std::cout << "points[" << i << "]: " << points[i] << std::endl;
  std::cout << std::endl;

  float2 centroid = compute_centroid(points);

  // TODO move these quadrants to the GPU using thrust::device_vector
  std::vector<int> quadrants(points.size());
  classify(points, centroid, quadrants);

  // TODO move these counts to the GPU using thrust::device_vector
  std::vector<int> counts_per_quadrant(4);
  count_points_in_quadrants(points, quadrants, counts_per_quadrant);

  std::cout << "Per-quadrant counts:" << std::endl;
  std::cout << "  Bottom-left : " << counts_per_quadrant[0] << " points" << std::endl;
  std::cout << "  Bottom-right: " << counts_per_quadrant[1] << " points" << std::endl;
  std::cout << "  Top-left    : " << counts_per_quadrant[2] << " points" << std::endl;
  std::cout << "  Top-right   : " << counts_per_quadrant[3] << " points" << std::endl;
  std::cout << std::endl;

  for(int i = 0; i < points.size(); ++i)
    std::cout << "points[" << i << "]: " << points[i] << std::endl;
  std::cout << std::endl;
}

